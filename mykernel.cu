#include "hip/hip_runtime.h"
#include <complex>
#include <hip/hip_complex.h>
#include "mykernel.h"

namespace dftfe {
    namespace {
        __device__ double Mult(double x,
                               double y) {
            return x * y;
        }

        __device__ hipDoubleComplex Mult(hipDoubleComplex x,
                                        hipDoubleComplex y) {
            return hipCmul(x, y);
        }

        template<typename NumType>
        __global__
        void saxpy_kernel(int n,
                          NumType a,
                          NumType *x,
                          NumType *y) {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            if (i < n) y[i] = Mult(a, x[i]);
        }

    }

    template<typename NumType>
    void saxpy(int n,
               NumType a,
               NumType *x,
               NumType *y) {
        saxpy_kernel<<<1, 1>>>(n,
                               a,
                               x,
                               y);
    }

    template void
    saxpy(int n,
          double a,
          double *x,
          double *y);

    template void
    saxpy(int n,
          hipDoubleComplex a,
          hipDoubleComplex *x,
          hipDoubleComplex *y);
}
